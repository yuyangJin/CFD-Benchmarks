#include <cstdio>
#include <vector>
using std::vector;

#include "eno.h"
#include "util.cuh"

timer func_timer;
void ENO_Solver(double ***U, double ***U1, double ***U2, double ***F, double ***Fp, double ***Fd, double ***F_p,
                double ***F_d, double ***F_, double ***G, double ***Gp, double ***Gd, double ***G_p, double ***G_d,
                double ***G_, double ****LAMDA_, double ****q3p, double ****q3d, double dx, double dy, double &dt) {
  bound(U, dx, dy);
  LF_x(U, LAMDA_, F, Fp, Fd);
  func_timer.start("ENO_x");
  ENO_x(U, F, Fp, Fd, F_p, F_d, F_, q3p, q3d, dx, dy, dt);
  func_timer.stop("ENO_x");
  bound(U, dx, dy);
  LF_y(U, LAMDA_, G, Gp, Gd);
  ENO_y(U, G, Gp, Gd, G_p, G_d, G_, q3p, q3d, dx, dy, dt);
  bound(U, dx, dy);
  LF_y(U, LAMDA_, G, Gp, Gd);
  ENO_y(U, G, Gp, Gd, G_p, G_d, G_, q3p, q3d, dx, dy, dt);
  bound(U, dx, dy);
  LF_x(U, LAMDA_, F, Fp, Fd);
  func_timer.start("ENO_x");
  ENO_x(U, F, Fp, Fd, F_p, F_d, F_, q3p, q3d, dx, dy, dt);
  func_timer.stop("ENO_x");
  bound(U, dx, dy);
}

void alloc(double **ptr, const vector<int> &dims, int depth) {
  if (depth + 1 >= dims.size()) return;
  for (int i = 0; i < dims[depth]; ++i) {
    hipMallocManaged(&(ptr[i]), dims[depth + 1] * sizeof(void *));
    alloc((double **)ptr[i], dims, depth + 1);
  }
}

double *alloc_nd(const vector<int> &dims) {
  double *output = nullptr;
  hipMallocManaged(&output, dims[0] * sizeof(void *));
  alloc((double **)output, dims, 0);
  return output;
}

int main(int argc, char **argv) {
  double dx, dy, dt = 0, T = 0;

  vector<int> shape1 = {Nx + 7, Ny + 7, 4};
  vector<int> shape2 = {Nx + 7, Ny + 7, 4, 4};
  vector<int> shape3 = {Nx + 7, Ny + 7, 4, 3};
  vector<int> shape4 = {Nx + 7, Ny + 7, 1};

  double ***U = (double ***)alloc_nd(shape1);
  double ***U_ = (double ***)alloc_nd(shape1);
  double ***Ut = (double ***)alloc_nd(shape1);
  double ***U1 = (double ***)alloc_nd(shape1);
  double ***U2 = (double ***)alloc_nd(shape1);
  double ***G = (double ***)alloc_nd(shape1);
  double ***G_ = (double ***)alloc_nd(shape1);
  double ***F = (double ***)alloc_nd(shape1);
  double ***F_ = (double ***)alloc_nd(shape1);
  double ***G_p = (double ***)alloc_nd(shape1);
  double ***G_d = (double ***)alloc_nd(shape1);
  double ***Gp = (double ***)alloc_nd(shape1);
  double ***Gd = (double ***)alloc_nd(shape1);
  double ***F_p = (double ***)alloc_nd(shape1);
  double ***F_d = (double ***)alloc_nd(shape1);
  double ***Fp = (double ***)alloc_nd(shape1);
  double ***Fd = (double ***)alloc_nd(shape1);

  double ****L_ = (double ****)alloc_nd(shape2);
  double ****R_ = (double ****)alloc_nd(shape2);
  double ****LAMDA_ = (double ****)alloc_nd(shape2);

  double ****q3p = (double ****)alloc_nd(shape3);
  double ****q3d = (double ****)alloc_nd(shape3);

  double ***a_ = (double ***)alloc_nd(shape4);

  initial(U, dx, dy);

  int n = 0;
  timeval s, e;
  gettimeofday(&s, NULL);
  while (T <= TT) {
    dt = CFL(U, dx, dy, ENOCFL);
    ENO_Solver(U, U1, U2, F, Fp, Fd, F_p, F_d, F_, G, Gp, Gd, F_p, F_d, F_, LAMDA_, q3p, q3d, dx, dy, dt);
    T += dt;
    n++;
    virtual_clear(U, dx, dy);
  }
  gettimeofday(&e, NULL);
  double ms = get_elapsed_time_ms(s, e);
  printf("ENO total time: %lf ms\n", ms);
  printf("ENO iter: %d\n", n);

  func_timer.show_all();

  if (argc == 2) {
    // result check
    printf("check dat file: %s\n", argv[1]);
    check(argv[1], U);
  }
  return 0;
}