#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
using std::vector;

#include "eno.cuh"
#include "util.cuh"

dim3 grid, block;
dim3 grid2, block2;
dim3 grid3, block3;
const int block_size = 32;
timer func_timer;

int offset1 = (Ny + 7) * 4;
int offset2 = 4;
double *U_cuda, *F_cuda, *Fp_cuda, *Fd_cuda, *F_p_cuda, *F_d_cuda, *F__cuda, *q3p_cuda, *q3d_cuda;
void ENO_Solver(double ***U, double ***U1, double ***U2, double ***F, double ***Fp, double ***Fd, double ***F_p,
                double ***F_d, double ***F_, double ***G, double ***Gp, double ***Gd, double ***G_p, double ***G_d,
                double ***G_, double ****LAMDA_, double ****q3p, double ****q3d, double dx, double dy, double &dt) {
  bound(U, dx, dy);
  LF_x(U, LAMDA_, F, Fp, Fd);
  func_timer.start("ENO_x");
  ENO_x(U, F, Fp, Fd, F_p, F_d, F_, q3p, q3d, dx, dy, dt);
  func_timer.stop("ENO_x");

  func_timer.start("ENO_x_cuda");
  eno_x_cuda<<<grid, block>>>(U_cuda, F_cuda, Fp_cuda, Fd_cuda, F_p_cuda, F_d_cuda, F__cuda, q3p_cuda, q3d_cuda, dx, dy,
                              dt, offset1, offset2);

  // __global__ void eno_x_cuda_2(double* U, double* F_, double dy, double r, int offset1, int offset2) {
  grid2.x = (Nx + 1 + 31) / 32;
  grid2.y = (int(0.5 / dy) + 1 + 31) / 32;
  eno_x_cuda_2<<<grid, block>>>(U_cuda, F_cuda, dy, 1.f, offset1, offset2);

  // __global__ void eno_x_cuda_3(double* U, double* F_, double dx, double dy, double r, int offset1, int offset2) {
  grid3.x = ((2.0 / dx - 1.0 / dx) + 1 + 31) / 32;
  grid3.y = (Ny - int(0.5 / dy) + 1 + 31) / 32;
  eno_x_cuda_3<<<grid, block>>>(U_cuda, F_cuda, dx, dy, 1.f, offset1, offset2);
  hipDeviceSynchronize();
  func_timer.stop("ENO_x_cuda");

  bound(U, dx, dy);
  LF_y(U, LAMDA_, G, Gp, Gd);
  ENO_y(U, G, Gp, Gd, G_p, G_d, G_, q3p, q3d, dx, dy, dt);
  bound(U, dx, dy);
  LF_y(U, LAMDA_, G, Gp, Gd);
  ENO_y(U, G, Gp, Gd, G_p, G_d, G_, q3p, q3d, dx, dy, dt);
  bound(U, dx, dy);
  LF_x(U, LAMDA_, F, Fp, Fd);
  func_timer.start("ENO_x");
  ENO_x(U, F, Fp, Fd, F_p, F_d, F_, q3p, q3d, dx, dy, dt);
  func_timer.stop("ENO_x");
  bound(U, dx, dy);
}

void alloc(double **ptr, const vector<int> &dims, int depth) {
  if (depth + 1 >= dims.size()) return;
  for (int i = 0; i < dims[depth]; ++i) {
    hipMallocManaged(&(ptr[i]), dims[depth + 1] * sizeof(void *));
    alloc((double **)ptr[i], dims, depth + 1);
  }
}

double *alloc_nd(const vector<int> &dims) {
  double *output = nullptr;
  hipMallocManaged(&output, dims[0] * sizeof(void *));
  alloc((double **)output, dims, 0);
  return output;
}

int main(int argc, char **argv) {
  double dx, dy, dt = 0, T = 0;

  vector<int> shape1 = {Nx + 7, Ny + 7, 4};
  int shape1_ = (Nx + 7) * (Ny + 7) * 4;
  vector<int> shape2 = {Nx + 7, Ny + 7, 4, 4};
  int shape2_ = (Nx + 7) * (Ny + 7) * 4 * 4;
  vector<int> shape3 = {Nx + 7, Ny + 7, 4, 3};
  int shape3_ = (Nx + 7) * (Ny + 7) * 4 * 3;
  vector<int> shape4 = {Nx + 7, Ny + 7, 1};
  int shape4_ = (Nx + 7) * (Ny + 7) * 1;

  double ***U = (double ***)alloc_nd(shape1);
  double ***U_ = (double ***)alloc_nd(shape1);
  double ***Ut = (double ***)alloc_nd(shape1);
  double ***U1 = (double ***)alloc_nd(shape1);
  double ***U2 = (double ***)alloc_nd(shape1);
  double ***G = (double ***)alloc_nd(shape1);
  double ***G_ = (double ***)alloc_nd(shape1);
  double ***F = (double ***)alloc_nd(shape1);
  double ***F_ = (double ***)alloc_nd(shape1);
  double ***G_p = (double ***)alloc_nd(shape1);
  double ***G_d = (double ***)alloc_nd(shape1);
  double ***Gp = (double ***)alloc_nd(shape1);
  double ***Gd = (double ***)alloc_nd(shape1);
  double ***F_p = (double ***)alloc_nd(shape1);
  double ***F_d = (double ***)alloc_nd(shape1);
  double ***Fp = (double ***)alloc_nd(shape1);
  double ***Fd = (double ***)alloc_nd(shape1);

  double ****L_ = (double ****)alloc_nd(shape2);
  double ****R_ = (double ****)alloc_nd(shape2);
  double ****LAMDA_ = (double ****)alloc_nd(shape2);

  double ****q3p = (double ****)alloc_nd(shape3);
  double ****q3d = (double ****)alloc_nd(shape3);

  double ***a_ = (double ***)alloc_nd(shape4);

  hipMalloc((void **)&U_cuda, shape1_ * sizeof(double));
  hipMalloc((void **)&F_cuda, shape1_ * sizeof(double));
  hipMalloc((void **)&Fp_cuda, shape1_ * sizeof(double));
  hipMalloc((void **)&Fd_cuda, shape1_ * sizeof(double));
  hipMalloc((void **)&F_p_cuda, shape1_ * sizeof(double));
  hipMalloc((void **)&F_d_cuda, shape1_ * sizeof(double));
  hipMalloc((void **)&F__cuda, shape1_ * sizeof(double));
  hipMalloc((void **)&q3p_cuda, shape3_ * sizeof(double));
  hipMalloc((void **)&q3d_cuda, shape3_ * sizeof(double));

  const int blocksize = 32;
  grid.x = (Nx + 2 + 31) / blocksize;
  grid.y = (Ny + 2 + 31) / blocksize;
  block.x = blocksize;
  block.y = blocksize;
  block2.x = blocksize;
  block2.y = blocksize;
  block3.x = blocksize;
  block3.y = blocksize;

  // int shape1_ = (Nx + 7) * (Ny + 7) * 4;

  for (int iter = 0; iter < 10; ++iter) {
    initial(U, dx, dy);
    int n = 0;
    timeval s, e;
    gettimeofday(&s, NULL);
    while (T <= TT) {
      dt = CFL(U, dx, dy, ENOCFL);
      ENO_Solver(U, U1, U2, F, Fp, Fd, F_p, F_d, F_, G, Gp, Gd, F_p, F_d, F_, LAMDA_, q3p, q3d, dx, dy, dt);
      T += dt;
      n++;
      virtual_clear(U, dx, dy);
    }
    gettimeofday(&e, NULL);
    double ms = get_elapsed_time_ms(s, e);
    printf("ENO total time: %lf ms\n", ms);
    printf("ENO iter: %d\n", n);
  }

  func_timer.show_all();

  if (argc == 2) {
    // result check
    printf("check dat file: %s\n", argv[1]);
    check(argv[1], U);
  }
  return 0;
}