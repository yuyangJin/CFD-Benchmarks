#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
using std::vector;

#include "eno.cuh"
#include "util.cuh"
// #define CUDAV2

dim3 grid, block;
dim3 grid2, block2;
dim3 grid3, block3;
const int blocksize = 32;
#ifdef CUDAV2
const int blocksize_Y = 8;
#else
const int blocksize_Y = 32;
#endif

timer func_timer;

int offset1 = (Ny + 7) * 4;
int offset2 = 4;
double *U_cuda, *F_cuda, *Fp_cuda, *Fd_cuda, *F_p_cuda, *F_d_cuda, *F__cuda, *q3p_cuda, *q3d_cuda;
void ENO_Solver(Tensor &U, Tensor &U1, Tensor &U2, Tensor &F, Tensor &Fp, Tensor &Fd, Tensor &F_p, Tensor &F_d,
                Tensor &F_, Tensor &G, Tensor &Gp, Tensor &Gd, Tensor &G_p, Tensor &G_d, Tensor &G_, Tensor &LAMDA_,
                Tensor &q3p, Tensor &q3d, double dx, double dy, double &dt) {
  bound_Tensor(U, dx, dy);
  LF_x_Tensor(U, LAMDA_, F, Fp, Fd);
  func_timer.start("ENO_x");
  ENO_x_Tensor(U, F, Fp, Fd, F_p, F_d, F_, q3p, q3d, dx, dy, dt);
  func_timer.stop("ENO_x");

  func_timer.start("ENO_x_cuda");
#ifdef CUDAV2
  eno_x_cuda_v2<<<grid, block>>>(U_cuda, F_cuda, Fp_cuda, Fd_cuda, F_p_cuda, F_d_cuda, F__cuda, q3p_cuda, q3d_cuda, dx,
                                 dy, dt, offset1, offset2);
#else
  eno_x_cuda<<<grid, block>>>(U_cuda, F_cuda, Fp_cuda, Fd_cuda, F_p_cuda, F_d_cuda, F__cuda, q3p_cuda, q3d_cuda, dx, dy,
                              dt, offset1, offset2);
#endif

  // __global__ void eno_x_cuda_2(double* U, double* F_, double dy, double r, int offset1, int offset2) {
  grid2.x = (Nx + 1 + blocksize - 1) / blocksize;
  grid2.y = (int(0.5 / dy) + 1 + blocksize_Y - 1) / blocksize_Y;
#ifdef CUDAV2
  eno_x_cuda_2_v2<<<grid2, block2>>>(U_cuda, F_cuda, dy, 1.f, offset1, offset2);
#else
  eno_x_cuda_2<<<grid2, block2>>>(U_cuda, F_cuda, dy, 1.f, offset1, offset2);
#endif

  // __global__ void eno_x_cuda_3(double* U, double* F_, double dx, double dy, double r, int offset1, int offset2) {
  grid3.x = ((2.0 / dx - 1.0 / dx) + 1 + blocksize - 1) / blocksize;
  grid3.y = (Ny - int(0.5 / dy) + 1 + blocksize_Y - 1) / blocksize_Y;
#ifdef CUDAV2
  eno_x_cuda_3_v2<<<grid3, block3>>>(U_cuda, F_cuda, dx, dy, 1.f, offset1, offset2);
#else
  eno_x_cuda_3<<<grid3, block3>>>(U_cuda, F_cuda, dx, dy, 1.f, offset1, offset2);
#endif
  hipDeviceSynchronize();
  func_timer.stop("ENO_x_cuda");

  bound_Tensor(U, dx, dy);
  LF_y_Tensor(U, LAMDA_, G, Gp, Gd);
  ENO_y_Tensor(U, G, Gp, Gd, G_p, G_d, G_, q3p, q3d, dx, dy, dt);
  bound_Tensor(U, dx, dy);
  LF_y_Tensor(U, LAMDA_, G, Gp, Gd);
  ENO_y_Tensor(U, G, Gp, Gd, G_p, G_d, G_, q3p, q3d, dx, dy, dt);
  bound_Tensor(U, dx, dy);
  LF_x_Tensor(U, LAMDA_, F, Fp, Fd);
  // func_timer.start("ENO_x");
  ENO_x_Tensor(U, F, Fp, Fd, F_p, F_d, F_, q3p, q3d, dx, dy, dt);
  // func_timer.stop("ENO_x");
  bound_Tensor(U, dx, dy);
}

void alloc(double **ptr, const vector<int> &dims, int depth) {
  if (depth + 1 >= dims.size()) return;
  for (int i = 0; i < dims[depth]; ++i) {
    ptr[i] = (double *)malloc(dims[depth + 1] * sizeof(void *));
    // hipMallocManaged(&(ptr[i]), dims[depth + 1] * sizeof(void *));
    alloc((double **)ptr[i], dims, depth + 1);
  }
}

double *alloc_nd(const vector<int> &dims) {
  double *output = nullptr;
  output = (double *)malloc(dims[0] * sizeof(void *));
  // hipMallocManaged(&output, dims[0] * sizeof(void *));
  alloc((double **)output, dims, 0);
  return output;
}

int main(int argc, char **argv) {
  double dx, dy, dt = 0, T = 0;

  vector<int> shape1 = {Nx + 7, Ny + 7, 4};
  int shape1_ = (Nx + 7) * (Ny + 7) * 4;
  vector<int> shape2 = {Nx + 7, Ny + 7, 4, 4};
  int shape2_ = (Nx + 7) * (Ny + 7) * 4 * 4;
  vector<int> shape3 = {Nx + 7, Ny + 7, 4, 3};
  int shape3_ = (Nx + 7) * (Ny + 7) * 4 * 3;
  vector<int> shape4 = {Nx + 7, Ny + 7, 1};
  int shape4_ = (Nx + 7) * (Ny + 7) * 1;

  Tensor U(shape1, shape1_);
  Tensor U_(shape1, shape1_);
  Tensor Ut(shape1, shape1_);
  Tensor U1(shape1, shape1_);
  Tensor U2(shape1, shape1_);
  Tensor G(shape1, shape1_);
  Tensor G_(shape1, shape1_);
  Tensor F(shape1, shape1_);
  Tensor F_(shape1, shape1_);
  Tensor G_p(shape1, shape1_);
  Tensor G_d(shape1, shape1_);
  Tensor Gp(shape1, shape1_);
  Tensor Gd(shape1, shape1_);
  Tensor F_p(shape1, shape1_);
  Tensor F_d(shape1, shape1_);
  Tensor Fp(shape1, shape1_);
  Tensor Fd(shape1, shape1_);
  // double ***U = (double ***)alloc_nd(shape1);
  // double ***U_ = (double ***)alloc_nd(shape1);
  // double ***Ut = (double ***)alloc_nd(shape1);
  // double ***U1 = (double ***)alloc_nd(shape1);
  // double ***U2 = (double ***)alloc_nd(shape1);
  // double ***G = (double ***)alloc_nd(shape1);
  // double ***G_ = (double ***)alloc_nd(shape1);
  // double ***F = (double ***)alloc_nd(shape1);
  // double ***F_ = (double ***)alloc_nd(shape1);
  // double ***G_p = (double ***)alloc_nd(shape1);
  // double ***G_d = (double ***)alloc_nd(shape1);
  // double ***Gp = (double ***)alloc_nd(shape1);
  // double ***Gd = (double ***)alloc_nd(shape1);
  // double ***F_p = (double ***)alloc_nd(shape1);
  // double ***F_d = (double ***)alloc_nd(shape1);
  // double ***Fp = (double ***)alloc_nd(shape1);
  // double ***Fd = (double ***)alloc_nd(shape1);

  Tensor L_(shape2, shape2_);
  Tensor R_(shape2, shape2_);
  Tensor LAMDA_(shape2, shape2_);
  // double ****L_ = (double ****)alloc_nd(shape2);
  // double ****R_ = (double ****)alloc_nd(shape2);
  // double ****LAMDA_ = (double ****)alloc_nd(shape2);

  Tensor q3p(shape3, shape3_);
  Tensor q3d(shape3, shape3_);
  // double ****q3p = (double ****)alloc_nd(shape3);
  // double ****q3d = (double ****)alloc_nd(shape3);

  Tensor a_(shape4, shape4_);
  // double ***a_ = (double ***)alloc_nd(shape4);

  hipMalloc((void **)&U_cuda, shape1_ * sizeof(double));
  hipMalloc((void **)&F_cuda, shape1_ * sizeof(double));
  hipMalloc((void **)&Fp_cuda, shape1_ * sizeof(double));
  hipMalloc((void **)&Fd_cuda, shape1_ * sizeof(double));
  hipMalloc((void **)&F_p_cuda, shape1_ * sizeof(double));
  hipMalloc((void **)&F_d_cuda, shape1_ * sizeof(double));
  hipMalloc((void **)&F__cuda, shape1_ * sizeof(double));
  hipMalloc((void **)&q3p_cuda, shape3_ * sizeof(double));
  hipMalloc((void **)&q3d_cuda, shape3_ * sizeof(double));

  grid.x = (Nx + 2 + blocksize - 1) / blocksize;
  grid.y = (Ny + 2 + blocksize_Y - 1) / blocksize_Y;
  block.x = blocksize;
  block.y = blocksize_Y;
  block2.x = blocksize;
  block2.y = blocksize_Y;
  block3.x = blocksize;
  block3.y = blocksize_Y;
#ifdef CUDAV2
  block.z = 3;
  block2.z = 3;
  block3.z = 3;
#endif

  // int shape1_ = (Nx + 7) * (Ny + 7) * 4;

  initial_Tensor(U, dx, dy);
  int n = 0;
  timeval s, e;
  gettimeofday(&s, NULL);
  while (T <= TT) {
    dt = CFL_Tensor(U, dx, dy, ENOCFL);
    ENO_Solver(U, U1, U2, F, Fp, Fd, F_p, F_d, F_, G, Gp, Gd, F_p, F_d, F_, LAMDA_, q3p, q3d, dx, dy, dt);
    T += dt;
    n++;
    virtual_clear_Tensor(U, dx, dy);
  }
  gettimeofday(&e, NULL);
  double ms = get_elapsed_time_ms(s, e);
  printf("ENO total time: %lf ms\n", ms);
  printf("ENO iter: %d\n", n);

  func_timer.show_all();

  if (argc == 2) {
    // result check
    printf("check dat file: %s\n", argv[1]);
    check_Tensor(argv[1], U);
  }
  return 0;
}